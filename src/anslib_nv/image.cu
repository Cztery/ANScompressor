#include "hip/hip_runtime.h"
#include "common_nv.cuh"
#include "image.cuh"
#include "image.h"

namespace anslib {

ChunkingParams::ChunkingParams(const ImgInfo &info, size_t chunkWid)
    : squareChunkWid_(chunkWid) {
  isRightEdgeMisaligned_ = info.width_ % chunkWid;
  isBottomEdgeMisaligned_ = info.height_ % chunkWid;
  chunksCountHor_ = info.width_ / chunkWid;
  chunksCountVer_ = info.height_ / chunkWid;
  chunksPerPlaneCount_ = chunksCountHor_ * chunksCountVer_;
  totalChunksCount_ = chunksPerPlaneCount_ * info.numOfPlanes_;
  edgeChunkHei_ = squareChunkWid_ + info.height_ % squareChunkWid_;
  edgeChunkWid_ = squareChunkWid_ + info.width_ % squareChunkWid_;
  maxChunkSize_ = edgeChunkWid_ * edgeChunkHei_ * ansSymbolTypeSize;
}

ChunkingParams::ChunkingParams(const ImgInfo &info)
    : ChunkingParams(info, info.chunkWidth_) {};

ImageDev::ImageDev(const RawImage &ri) {
  printf("constructor called\n");
  imgInfo.width_ = ri.width_;
  imgInfo.height_ = ri.height_;
  imgInfo.numOfPlanes_ = ri.numOfPlanes_;
  imgInfo.numOfChunksPerPlane_ = ri.chunksPerPlaneCount();
  imgInfo.chunkWidth_ = ri.chunkWidth_;

  CHECK_CUDA_ERROR(hipMalloc(&rawChunks, ri.bytesSizeOfImage()));
  AnsSymbolType *tmp_chunksPtr = rawChunks;
  for (auto chunk : ri.dataPlanes_) {
    size_t chunkSizeBytes = chunk.size() * sizeof(AnsSymbolType);
    CHECK_CUDA_ERROR(hipMemcpy(tmp_chunksPtr, chunk.data(), chunkSizeBytes,
                                hipMemcpyHostToDevice));
    tmp_chunksPtr += chunk.size();
  }
}

ImageDev::ImageDev(const CompImage &ci) {
  printf("constructor called\n");
  imgInfo.width_ = ci.width_;
  imgInfo.height_ = ci.height_;
  imgInfo.numOfPlanes_ = ci.numOfPlanes_;
  imgInfo.numOfChunksPerPlane_ = ci.chunksPerPlaneCount();
  imgInfo.chunkWidth_ = ci.chunkWidth_;

  const size_t numOfChunksTotal =
      imgInfo.numOfPlanes_ * imgInfo.numOfChunksPerPlane_;
  hipMalloc(&chunkCounts, numOfChunksTotal * ansCountsArrSize);

  hipMalloc(&compChunksSizes, numOfChunksTotal * sizeof(size_t));
  size_t compChunksTotalSize = 0;
  std::vector<size_t> compChunksSizes_tmp;
  for (size_t i = 0; i < ci.compressedPlanes_.size(); ++i) {
    compChunksTotalSize += ci.compressedPlanes_.size();
    compChunksSizes_tmp.push_back(ci.compressedPlanes_.at(i).plane.size());
  }

  CHECK_CUDA_ERROR(hipMemcpy(compChunksSizes, compChunksSizes_tmp.data(),
                              compChunksSizes_tmp.size() * sizeof(size_t),
                              hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(
      hipMalloc(&compChunks, compChunksTotalSize * ansCompTypeSize));

  AnsCompType *compChunksPtr = compChunks;
  for (size_t i = 0; i < ci.compressedPlanes_.size(); ++i) {
    CHECK_CUDA_ERROR(hipMemcpy(
        compChunksPtr, ci.compressedPlanes_.at(i).plane.data(),
        ci.compressedPlanes_.at(i).plane.size(), hipMemcpyHostToDevice));
    compChunksPtr += compChunksSizes_tmp[i];
  }
}

ImageDev::~ImageDev() {
  printf("destructor called\n");

  // release device-side pointer arrays
  if (rawChunks) CHECK_CUDA_ERROR(hipFree(rawChunks));
  if (compChunks) CHECK_CUDA_ERROR(hipFree(compChunks));
  if (chunkCounts) CHECK_CUDA_ERROR(hipFree(chunkCounts));
  if (chunkCumul) CHECK_CUDA_ERROR(hipFree(chunkCumul));
  if (compChunksSizes) CHECK_CUDA_ERROR(hipFree(compChunksSizes));
}

__device__ void d_splitIntoChunks(AnsSymbolType *rawChunk,
                                  ChunkingParams cParams,
                                  ImageDev img) {
  const bool isRightEdge = cParams.isRightEdgeMisaligned_ &&
                           (cParams.chunksCountHor_ - 1 == blockIdx.x);
  const bool isBottomEdge = cParams.isBottomEdgeMisaligned_ &&
                            (cParams.chunksCountVer_ - 1 == blockIdx.y);

  const size_t chunkWid =
      isRightEdge ? cParams.edgeChunkWid_ : cParams.squareChunkWid_;
  const size_t chunkHei =
      isBottomEdge ? cParams.edgeChunkHei_ : cParams.squareChunkWid_;
  if (chunkHei <= threadIdx.y || chunkWid <= threadIdx.x) return;

  const size_t planeOffset = blockIdx.z * img.imgInfo.width_ * img.imgInfo.height_;
  const size_t xOffsetOfChunkIn = blockIdx.x * cParams.squareChunkWid_;
  const size_t yOffsetOfChunkIn = blockIdx.y * cParams.squareChunkWid_;
  const size_t inPlanePixIdx = (yOffsetOfChunkIn + threadIdx.x) * img.imgInfo.width_ +
                               xOffsetOfChunkIn + threadIdx.x;

  rawChunk[threadIdx.x + threadIdx.y * chunkWid] =
      img.rawChunks[planeOffset + inPlanePixIdx];
}

__device__ void d_joinChunks(AnsSymbolType *outPlanes, const AnsSymbolType *inChunks,
                             const ChunkingParams cParams, const ImgInfo imgInfo) {
  const bool isRightEdge = cParams.isRightEdgeMisaligned_ &&
                           (cParams.chunksCountHor_ - 1 == blockIdx.x);
  const bool isBottomEdge = cParams.isBottomEdgeMisaligned_ &&
                            (cParams.chunksCountVer_ - 1 == blockIdx.y);

  const size_t chunkWid =
      isRightEdge ? cParams.edgeChunkWid_ : cParams.squareChunkWid_;
  const size_t chunkHei =
      isBottomEdge ? cParams.edgeChunkHei_ : cParams.squareChunkWid_;
  if (chunkHei <= threadIdx.y || chunkWid <= threadIdx.x) return;

  const size_t chunkOffsetOut =
      blockIdx.y * ((cParams.chunksCountHor_ - 1) * cParams.squareChunkWid_ *
                        cParams.squareChunkWid_ +
                    cParams.squareChunkWid_ * cParams.edgeChunkWid_) +
      blockIdx.x * chunkHei * cParams.squareChunkWid_;
  const size_t chunkVerPosInPlane = threadIdx.y * cParams.squareChunkWid_;
  const size_t planeOffset = blockIdx.z * imgInfo.width_ * imgInfo.height_;
  const size_t xOffsetOfChunkOut = blockIdx.x * cParams.squareChunkWid_;
  const size_t yOffsetOfChunkOut = blockIdx.y * cParams.squareChunkWid_;
  const size_t outPlanePixIdx = (yOffsetOfChunkOut + threadIdx.y) * imgInfo.width_ +
                                xOffsetOfChunkOut + threadIdx.x;
  outPlanes[planeOffset + outPlanePixIdx] =
      inChunks[planeOffset + chunkOffsetOut + chunkVerPosInPlane];
}

__global__ void compressionPipeline(const ChunkingParams &cParams,
      ImageDev *imgInOut) {
  __shared__ uint8_t sChunkMem[];

  AnsSymbolType *sRawChunk = sChunkMem;
  AnsCountsType *sChunkCounts = sChunkMem + cParams.maxChunkSize_;
  AnsCountsType *sChunkCumul = sChunkCounts + ansCountsArrSize;
  AnsCompType *sCompressedChunk = (AnsCompType *)sChunkCumul + ansCumulArrSize;

  d_splitIntoChunks(sRawChunk, cParams, imgIn);
}

void ImageDev::runCompressionPipeline(size_t chunkWid) {
  const ChunkingParams cParams(imgInfo, chunkWid);

  // 2D block indices correspond to chunk location in plane
  dim3 grid(cParams.chunksCountHor_, cParams.chunksCountVer_,
            imgInfo.numOfPlanes_);
  // 2D thread indices correspond to x and y idx in chunk
  dim3 block(cParams.edgeChunkWid_, cParams.edgeChunkHei_);

  const size_t sharedMemSize =
      cParams.maxChunkSize_ * 2 + ansCountsArrSize + ansCumulArrSize;

  ImageDev *imgDevInOut = nullptr;
  CHECK_CUDA_ERROR(hipMalloc(&imgDevInOut, sizeof(ImageDev)));
  CHECK_CUDA_ERROR(hipMemcpy(imgDevInOut, this, sizeof(ImageDev),
  compressionPipeline<<<grid, block, sharedMemSize>>>(cParams, imgDevInOut);
  CHECK_CUDA_ERROR(hipMemcpy(this, imgDevInOut, sizeof(ImageDev),
                              hipMemcpyDeviceToHost));

  const size_t compDataSize = 0;
  for (int i = 0; i < cParams.chunksPerPlaneCount_ * imgInfo.numOfPlanes_;
       ++i) {
    compDataSize += compChunksSizes[i];
  }
  CHECK_CUDA_ERROR(hipMalloc(&compChunks, compDataSize));
}

__global__ void ImageDev::decompressionPipeline(const ChunkingParams &cParams,
      ImageDev *imgInOut) {
  __shared__ uint8_t sChunkMem[];
  AnsSymbolType *sRawChunk = sChunkMem;
  AnsCountsType *sChunkCounts = sChunkMem + cParams.maxChunkSize_;
  AnsCountsType *sChunkCumul = sChunkCounts + ansCountsArrSize;
  AnsCompType *sCompressedChunk = sChunkCumul + ansCumulArrSize;
  // decompress

  d_joinChunks(rawChunks, sRawChunk, cParams, imgInfo);
}

void ImageDev::runDecompressionPipeline() { 
  const ChunkingParams cParams(imgInfo);

  // 2D block indices correspond to chunk location in plane
  dim3 grid(cParams.chunksCountHor_, cParams.chunksCountVer_,
            imgInfo.numOfPlanes_);
  // 2D thread indices correspond to x and y idx in chunk
  dim3 block(cParams.edgeChunkWid_, cParams.edgeChunkHei_);

  const size_t sharedMemSize =
      cParams.maxChunkSize_ * 2 + ansCountsArrSize + ansCumulArrSize;

  ImageDev *imgDevInOut = nullptr;
  CHECK_CUDA_ERROR(hipMalloc(&imgDevInOut, sizeof(ImageDev)));
  CHECK_CUDA_ERROR(hipMemcpy(imgDevInOut, this, sizeof(ImageDev),
                              hipMemcpyHostToDevice));
  decompressionPipeline<<<grid, block, sharedMemSize>>>(cParams, imgDevInOut);
  CHECK_CUDA_ERROR(hipMemcpy(this, imgDevInOut, sizeof(ImageDev),
                              hipMemcpyDeviceToHost));
}

const std::vector<AnsSymbolType> ImageDev::getPlane(size_t idx) {
  if (idx >= (imgInfo.numOfPlanes_ * imgInfo.numOfChunksPerPlane_)) {
    return {};
  }
  const size_t planeSize =
      imgInfo.width_ * imgInfo.height_ * sizeof(AnsSymbolType);
  AnsSymbolType *planeTmp = (AnsSymbolType *)malloc(planeSize);
  CHECK_CUDA_ERROR(hipMemcpy(
      planeTmp, rawChunks + (idx * (imgInfo.width_ * imgInfo.height_)),
      planeSize, hipMemcpyDeviceToHost));
  std::vector<AnsSymbolType> vec_tmp(
      planeTmp, planeTmp + (imgInfo.height_ * imgInfo.width_));
  return vec_tmp;
}

}  // namespace anslib