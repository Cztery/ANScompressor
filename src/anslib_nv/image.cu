#include "hip/hip_runtime.h"
#include <exception>

#include "common_nv.cuh"
#include "image.cuh"
#include "image.h"

namespace anslib {

ChunkingParams::ChunkingParams(const ImgInfo &info, size_t chunkWid)
    : squareChunkWid_(chunkWid) {
  isRightEdgeMisaligned_ = info.width_ % chunkWid;
  isBottomEdgeMisaligned_ = info.height_ % chunkWid;
  chunksCountHor_ = (info.width_ + chunkWid - 1) / chunkWid;
  chunksCountVer_ = (info.height_ + chunkWid - 1) / chunkWid;
  chunksPerPlaneCount_ = chunksCountHor_ * chunksCountVer_;
  totalChunksCount_ = chunksPerPlaneCount_ * info.numOfPlanes_;
  edgeChunkWid_ = info.width_ % squareChunkWid_;
  edgeChunkHei_ = info.height_ % squareChunkWid_;
}

ChunkingParams::ChunkingParams(const ImgInfo &info)
    : ChunkingParams(info, info.chunkWidth_){};

ImageDev::ImageDev(const RawImage &ri) {
  printf("constructor called\n");
  imgInfo.width_ = ri.width_;
  imgInfo.height_ = ri.height_;
  imgInfo.numOfPlanes_ = ri.numOfPlanes_;
  imgInfo.numOfChunksPerPlane_ = ri.chunksPerPlaneCount();
  imgInfo.chunkWidth_ = ri.chunkWidth_;

  CHECK_CUDA_ERROR(hipMalloc(&rawChunks_, ri.bytesSizeOfImage()));
  AnsSymbolType *tmp_chunksPtr = rawChunks_;
  for (auto chunk : ri.dataPlanes_) {
    size_t chunkSizeBytes = chunk.size() * sizeof(AnsSymbolType);
    CHECK_CUDA_ERROR(hipMemcpy(tmp_chunksPtr, chunk.data(), chunkSizeBytes,
                                hipMemcpyHostToDevice));
    tmp_chunksPtr += chunk.size();
  }
}

ImageDev::ImageDev(const CompImage &ci) {
  printf("constructor called\n");
  imgInfo.width_ = ci.width_;
  imgInfo.height_ = ci.height_;
  imgInfo.numOfPlanes_ = ci.numOfPlanes_;
  imgInfo.numOfChunksPerPlane_ = ci.chunksPerPlaneCount();
  imgInfo.chunkWidth_ = ci.chunkWidth_;

  const size_t numOfChunksTotal =
      imgInfo.numOfPlanes_ * imgInfo.numOfChunksPerPlane_;
  hipMalloc(&chunkCounts_, numOfChunksTotal * ansCountsArrSize);

  hipMalloc(&compChunksSizes_, numOfChunksTotal * sizeof(size_t));
  size_t compChunksTotalSize = 0;
  std::vector<size_t> compChunksSizes_tmp;
  for (size_t i = 0; i < ci.compressedPlanes_.size(); ++i) {
    compChunksTotalSize += ci.compressedPlanes_.size();
    compChunksSizes_tmp.push_back(ci.compressedPlanes_.at(i).plane.size());
  }

  CHECK_CUDA_ERROR(hipMemcpy(compChunksSizes_, compChunksSizes_tmp.data(),
                              compChunksSizes_tmp.size() * sizeof(size_t),
                              hipMemcpyHostToDevice));

  CHECK_CUDA_ERROR(
      hipMalloc(&compChunks_, compChunksTotalSize * ansCompTypeSize));

  AnsCompType *compChunksPtr = compChunks_;
  for (size_t i = 0; i < ci.compressedPlanes_.size(); ++i) {
    CHECK_CUDA_ERROR(hipMemcpy(
        compChunksPtr, ci.compressedPlanes_.at(i).plane.data(),
        ci.compressedPlanes_.at(i).plane.size(), hipMemcpyHostToDevice));
    compChunksPtr += compChunksSizes_tmp[i];
  }
}

ImageDev::~ImageDev() {
  printf("destructor called\n");

  // release device-side pointer arrays
  if (rawChunks_) CHECK_CUDA_ERROR(hipFree(rawChunks_));
  rawChunks_ = nullptr;
  if (compChunks_) CHECK_CUDA_ERROR(hipFree(compChunks_));
  if (chunkCounts_) CHECK_CUDA_ERROR(hipFree(chunkCounts_));
  if (chunkCumul_) CHECK_CUDA_ERROR(hipFree(chunkCumul_));
  if (compChunksSizes_) CHECK_CUDA_ERROR(hipFree(compChunksSizes_));
}

__device__ void d_splitIntoChunks(AnsSymbolType *rawChunk,
                                  ChunkingParams cParams, ImageDev *img) {
  const bool isRightEdge = cParams.isRightEdgeMisaligned_ &&
                           (cParams.chunksCountHor_ - 1 == blockIdx.x);
  const bool isBottomEdge = cParams.isBottomEdgeMisaligned_ &&
                            (cParams.chunksCountVer_ - 1 == blockIdx.y);

  const size_t chunkWid =
      isRightEdge ? cParams.edgeChunkWid_ : cParams.squareChunkWid_;
  const size_t chunkHei =
      isBottomEdge ? cParams.edgeChunkHei_ : cParams.squareChunkWid_;
  if (chunkHei <= threadIdx.y || chunkWid <= threadIdx.x) return;

  const size_t planeOffset =
      blockIdx.z * img->imgInfo.width_ * img->imgInfo.height_;
  const size_t xOffsetOfChunkIn = blockIdx.x * cParams.squareChunkWid_;
  const size_t yOffsetOfChunkIn = blockIdx.y * cParams.squareChunkWid_;
  const size_t inPlanePixIdx =
      (yOffsetOfChunkIn + threadIdx.y) * img->imgInfo.width_ + xOffsetOfChunkIn;
  __syncthreads();
  if (threadIdx.x == 0 && threadIdx.y < chunkHei) {
    memcpy(&rawChunk[threadIdx.y * chunkWid],
           &img->rawChunks_[planeOffset + inPlanePixIdx],
           sizeof(anslib::AnsSymbolType) * chunkWid);
  }
  // rawChunk[threadIdx.x + threadIdx.y * chunkWid] =
  //     img->rawChunks_[planeOffset + inPlanePixIdx];

  // set chunkWidth_ and numOfChunksPerPlane_, only one thread per whole image
  if (!blockIdx.x && !blockIdx.y && !blockIdx.z && !threadIdx.x &&
      !threadIdx.y && !threadIdx.z) {
    img->imgInfo.chunkWidth_ = cParams.squareChunkWid_;
    img->imgInfo.numOfChunksPerPlane_ = cParams.chunksPerPlaneCount_;
  }
}

__device__ void d_joinChunks(AnsSymbolType *outPlanes,
                             const AnsSymbolType *inChunk,
                             const ChunkingParams cParams, ImgInfo &imgInfo) {
  const bool isRightEdge = cParams.isRightEdgeMisaligned_ &&
                           (cParams.chunksCountHor_ - 1 == blockIdx.x);
  const bool isBottomEdge = cParams.isBottomEdgeMisaligned_ &&
                            (cParams.chunksCountVer_ - 1 == blockIdx.y);

  const size_t chunkWid =
      isRightEdge ? cParams.edgeChunkWid_ : cParams.squareChunkWid_;
  const size_t chunkHei =
      isBottomEdge ? cParams.edgeChunkHei_ : cParams.squareChunkWid_;
  if (chunkHei <= threadIdx.y || chunkWid <= threadIdx.x) return;

  const size_t chunkOffsetOut =
      blockIdx.y * ((cParams.chunksCountHor_ - 1) * cParams.squareChunkWid_ *
                        cParams.squareChunkWid_ +
                    cParams.squareChunkWid_ * cParams.edgeChunkWid_) +
      blockIdx.x * chunkHei * cParams.squareChunkWid_;
  const size_t chunkVerPosInPlane = threadIdx.y * cParams.squareChunkWid_;
  const size_t planeOffset = blockIdx.z * imgInfo.width_ * imgInfo.height_;
  const size_t xOffsetOfChunkOut = blockIdx.x * cParams.squareChunkWid_;
  const size_t yOffsetOfChunkOut = blockIdx.y * cParams.squareChunkWid_;
  const size_t outPlanePixIdx =
      (yOffsetOfChunkOut + threadIdx.y) * imgInfo.width_ + xOffsetOfChunkOut;

  if (threadIdx.x == 0 && threadIdx.y < chunkHei) {
    // copy one row per thread, hence only #0 threads in x axis
    memcpy(&outPlanes[planeOffset + outPlanePixIdx],
           &inChunk[threadIdx.y * chunkWid],
           sizeof(anslib::AnsSymbolType) * chunkWid);
  }

  imgInfo.chunkWidth_ = 0;
  // outPlanes[planeOffset + outPlanePixIdx] =
  //     inChunks[planeOffset + chunkOffsetOut + chunkVerPosInPlane];
}

__global__ void compressionPipeline(const ChunkingParams cParams,
                                    ImageDev *imgInOut) {
  extern __shared__ uint8_t sChunkMem[];

  AnsSymbolType *sRawChunk = sChunkMem;
  AnsCountsType *sChunkCounts =
      (AnsCountsType *)(sChunkMem +
                        cParams.squareChunkWid_ * cParams.squareChunkWid_);
  AnsCountsType *sChunkCumul = sChunkCounts + ansCountsArrSize;
  AnsCompType *sCompressedChunk = (AnsCompType *)sChunkCumul + ansCumulArrSize;

  d_splitIntoChunks(sRawChunk, cParams, imgInOut);
}

ImageDev ImageCompressor::compress(size_t chunkWid) {
  if (!chunkWid) {
    throw std::invalid_argument("Provided invalid chunk width (0). Aborting.");
  }
  const ChunkingParams cParams(img_.imgInfo, chunkWid);

  // 2D block indices correspond to chunk location in plane
  dim3 grid(cParams.chunksCountHor_, cParams.chunksCountVer_,
            img_.imgInfo.numOfPlanes_);
  // 2D thread indices correspond to x and y idx in chunk
  const int threadRows = 1024 / cParams.squareChunkWid_;
  dim3 block(threadRows, cParams.squareChunkWid_);

  const size_t sharedMemSize =
      cParams.squareChunkWid_ * cParams.squareChunkWid_ * 2 *
          sizeof(AnsSymbolType) +
      (ansCountsArrSize + ansCumulArrSize) * sizeof(AnsCountsType);
  compressionPipeline<<<grid, block, sharedMemSize>>>(cParams, imgNv_);
  CHECK_LAST_CUDA_ERROR();
  ImageDev compImg;
  CHECK_CUDA_ERROR(
      hipMemcpy(&compImg, imgNv_, sizeof(ImageDev), hipMemcpyDeviceToHost));
  size_t compDataSize = 0;
  for (int i = 0; i < cParams.chunksPerPlaneCount_ * img_.imgInfo.numOfPlanes_;
       ++i) {
    compDataSize += compImg.compChunksSizes_[i];
  }
  return compImg;
}

__global__ void decompressionPipeline(const ChunkingParams cParams,
                                      ImageDev *imgInOut) {
  extern __shared__ uint8_t sChunkMem[];
  AnsSymbolType *sRawChunk = sChunkMem;
  AnsCountsType *sChunkCounts =
      (AnsCountsType *)(sChunkMem +
                        cParams.squareChunkWid_ * cParams.squareChunkWid_);
  AnsCountsType *sChunkCumul = sChunkCounts + ansCountsArrSize;
  AnsCompType *sCompressedChunk =
      (AnsCompType *)(sChunkCumul + ansCumulArrSize);
  // decompress

  d_joinChunks(imgInOut->rawChunks_, sRawChunk, cParams, imgInOut->imgInfo);
}

ImageDev ImageCompressor::decompress() {
  const ChunkingParams cParams(img_.imgInfo);

  // 2D block indices correspond to chunk location in plane
  dim3 grid(cParams.chunksCountHor_, cParams.chunksCountVer_,
            img_.imgInfo.numOfPlanes_);
  // 2D thread indices correspond to x and y idx in chunk
  const int threadRows = 1024 / cParams.squareChunkWid_;
  dim3 block(threadRows, cParams.squareChunkWid_);

  const size_t sharedMemSize =
      cParams.squareChunkWid_ * cParams.squareChunkWid_ * 2 + ansCountsArrSize +
      ansCumulArrSize;

  decompressionPipeline<<<grid, block, sharedMemSize>>>(cParams, imgNv_);
  CHECK_LAST_CUDA_ERROR();
  RawImage rawImg;
  CHECK_CUDA_ERROR(
      hipMemcpy(&rawImg, imgNv_, sizeof(RawImage), hipMemcpyDeviceToHost));
  return rawImg;
}

const std::vector<AnsSymbolType> ImageDev::getPlane(size_t idx) {
  if (idx >= (imgInfo.numOfPlanes_ * imgInfo.numOfChunksPerPlane_)) {
    return {};
  }
  const size_t planeSize =
      imgInfo.width_ * imgInfo.height_ * sizeof(AnsSymbolType);
  AnsSymbolType *planeTmp = (AnsSymbolType *)malloc(planeSize);
  CHECK_CUDA_ERROR(hipMemcpy(
      planeTmp, rawChunks_ + (idx * (imgInfo.width_ * imgInfo.height_)),
      planeSize, hipMemcpyDeviceToHost));
  std::vector<AnsSymbolType> vec_tmp(
      planeTmp, planeTmp + (imgInfo.height_ * imgInfo.width_));
  return vec_tmp;
}

ImageCompressor::ImageCompressor(const RawImage &ri) : img_(ri) {
  CHECK_CUDA_ERROR(hipMalloc(&imgNv_, sizeof(ImageDev)));
  CHECK_CUDA_ERROR(
      hipMemcpy(imgNv_, &img_, sizeof(ImageDev), hipMemcpyHostToDevice));
}

ImageCompressor::ImageCompressor(const CompImage &ci) : img_(ci) {
  CHECK_CUDA_ERROR(hipMalloc(&imgNv_, sizeof(ImageDev)));
  CHECK_CUDA_ERROR(
      hipMemcpy(imgNv_, &img_, sizeof(ImageDev), hipMemcpyHostToDevice));
}

ImageCompressor::~ImageCompressor() {
  if (imgNv_) hipFree(imgNv_);
}

}  // namespace anslib