#include <iostream>
#include <vector>
#include "bmplib.h"
#include "ppmlib.h"
#include "common.h"
#include "compressor.h"
#include "image.h"
#include "benchlib.h"

#include <cstdio>

struct ImgInfo {
  size_t width_;
  size_t height_;
  size_t numOfPlanes_;
  size_t numOfChunksPerPlane_;
  size_t chunkWidth_;
};

struct ImageDev {
  ImgInfo *info;
  anslib::AnsSymbol ***rawChunks;
  anslib::AnsCountsType ***chunkCounts;
  anslib::AnsCountsType ***chunkCumul;
  uint8_t ***compChunks;
  
  ImageDev(const anslib::RawImage &ri);
  ImageDev(const anslib::CompImage &ci);
  ~ImageDev();
};

ImageDev::ImageDev(const anslib::RawImage &ri) {
  printf("constructor called\n");

  hipMalloc((void**)info, sizeof(ImgInfo));
  ImgInfo tmpInfo{ri.width_, ri.height_, ri.numOfPlanes_, ri.chunksPerPlaneCount(), ri.chunkWidth_};
  hipMemcpy(info, &tmpInfo, sizeof(ImgInfo), hipMemcpyHostToDevice);

  hipMalloc((void**)rawChunks, ri.dataPlanes_.size());
  for(size_t chunkIdx = 0; chunkIdx < ri.dataPlanes_.size(); ++chunkIdx) {
    // chunk sizes may slightly vary for the chunks located at the edges of a plane
    const size_t chunkSize = ri.dataPlanes_.at(chunkIdx).size(); 
    hipMalloc((void**)rawChunks[chunkIdx], chunkSize);
    hipMalloc((void**)compChunks[chunkIdx], chunkSize);
    hipMalloc((void**)chunkCounts[chunkIdx], anslib::ansCountsSize);
  }
}

ImageDev::ImageDev(const anslib::CompImage &ci) {
  printf("constructor called\n");
  
  hipMalloc((void**)info, sizeof(ImgInfo));
  ImgInfo tmpInfo = {ci.width_, ci.height_, ci.numOfPlanes_, ci.chunksPerPlaneCount(), ci.chunkWidth_};
  hipMemcpy(info, &tmpInfo, sizeof(ImgInfo), hipMemcpyHostToDevice);
  
  hipMalloc((void**)compChunks, ci.compressedPlanes_.size());
  for(size_t chunkIdx = 0; chunkIdx < ci.compressedPlanes_.size(); ++chunkIdx) {
    // chunk sizes will vary for all compressed chunks
    const size_t chunkSize = ci.compressedPlanes_.at(chunkIdx).plane.size();

    hipMalloc((void**)compChunks[chunkIdx], chunkSize);
    hipMalloc((void**)chunkCounts[chunkIdx], anslib::ansCountsSize);
    hipMalloc((void**)rawChunks[chunkIdx], ci.compressedPlanes_.at(chunkIdx).rawPlaneSize);
    hipMemcpy(compChunks[chunkIdx], ci.compressedPlanes_.at(chunkIdx).plane.data(),
      chunkSize, hipMemcpyHostToDevice);
    hipMemcpy(chunkCounts[chunkIdx], ci.compressedPlanes_.at(chunkIdx).counts.data(),
      anslib::ansCountsSize, hipMemcpyHostToDevice);
  }
}

ImageDev::~ImageDev() {
  printf("destructor called\n");
  
  for(size_t chunkIdx = 0; chunkIdx < info->numOfChunksPerPlane_ * info->numOfPlanes_; ++chunkIdx) {
    hipFree(rawChunks[chunkIdx]);
    hipFree(compChunks[chunkIdx]);
    hipFree(chunkCounts[chunkIdx]);
    hipFree(chunkCumul[chunkIdx]);
  }
  hipFree(rawChunks);
  hipFree(compChunks);
  hipFree(chunkCounts);
  hipFree(chunkCumul);
  hipFree(info);
}

// struct ImageDev *copyRawImgToDevice(const anslib::RawImage &inImg) {
//   struct ImageDev *rawImgHandleHost(inImg);
//   struct ImageDev *rawImgHandleDevice;
//   hipMalloc((void**)rawImgHandleDevice, sizeof(struct ImageDev));
//   hipMemcpy(rawImgHandleDevice, rawImgHandleHost, hipMemcpyHostToDevice);
// }

// void copyCompImgFromDevice(anslib::CompImage &outImg, anslib::) {
//   struct ImageDev *compImgDev; // chce comp i metadata, nie chcę raw
//   hipMemcpy()
// }

// hipError_t freeRawImageDev(struct ImageDev *RIptr) {
//   hipFree();
// }

// hipArray_t x;
// void compressImageParallel(const anslib::RawImage &inImg, anslib::CompImage &outImg) {
//   RawImageCu *copyRawImgToDevice(inImg);                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                         
   
// }

int main(int argc, char *argv[]) {
  std::vector<FileStats> encodeStats;
  std::vector<std::string> testImgs;
  listAllImgsInDir(CMAKE_SOURCE_DIR "/test_images/PHOTO_CD_KODAK/BMP_IMAGES/",
                   ".bmp", testImgs);
  for (auto imgPath : testImgs) {
    for (uint32_t chunk_size = 0; chunk_size <= 512; chunk_size += 64) {
      anslib::RawImage rawImg = FileStats::getTestImg(imgPath);
      
      /// TODO: parallelize
      rawImg.splitIntoChunks(chunk_size);
      anslib::CompImage compImg;
      anslib::AnsEncoder::compressImage(rawImg, compImg);
      {
        ImageDev compContext(rawImg);
        ImageDev decompContext(compImg);
      }
      
      /// TODO: make a 
      // anslib::AnsEncoder::compressImage(img, resultImg);
      // anslib::AnsDecoder::decompressImage(resultImg, img);

      // std::cout << "Processing " << imgPath.substr(imgPath.rfind('/') + 1) << " for chunk_size = " << chunk_size << '\n';
      // for (size_t i = 0; i < img.dataPlanes_.size(); ++i) {
      //   assert(img.dataPlanes_.at(i).size() == imgRef.dataPlanes_.at(i).size());
      //   assert(img.dataPlanes_.at(i) == imgRef.dataPlanes_.at(i));
      // }
      
      // FileStats fs(img, imgPath.substr(imgPath.rfind('/') + 1));
      // encodeStats.push_back(fs);
      // std::cout << encodeStats.back();
    }
}
}