#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include "bmplib.h"
#include "ppmlib.h"
#include "common.h"
#include "compressor.h"
#include "image.h"
#include "benchlib.h"

#include <cstdio>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, char const* const func, char const* const file,
           int const line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(char const* const file, int const line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

struct ImgInfo {
  size_t width_;
  size_t height_;
  size_t numOfPlanes_;
  size_t numOfChunksPerPlane_;
  size_t chunkWidth_;
};

struct ImageDev {
  ImgInfo *h_info, *d_info;
  anslib::AnsSymbolType **rawChunks;
  anslib::AnsCountsType **chunkCounts;
  anslib::AnsCountsType **chunkCumul;
  anslib::AnsCompType **compChunks;

  anslib::AnsSymbolType **h_rawChunks;
  anslib::AnsCountsType **h_chunkCounts;
  anslib::AnsCountsType **h_chunkCumul;
  anslib::AnsCompType **h_compChunks;
  
  ImageDev(const anslib::RawImage &ri);
  ImageDev(const anslib::CompImage &ci);
  ~ImageDev();
};

ImageDev::ImageDev(const anslib::RawImage &ri) {
  printf("constructor called\n");

  CHECK_CUDA_ERROR(hipMalloc((void**)&d_info, sizeof(ImgInfo)));
  h_info = (ImgInfo *)malloc(sizeof(struct ImgInfo));
  ImgInfo tmpInfo = {ri.width_, ri.height_, ri.numOfPlanes_, ri.chunksPerPlaneCount(), ri.chunkWidth_};
  memcpy(h_info, &tmpInfo, sizeof(ImgInfo));
  CHECK_CUDA_ERROR(hipMemcpy(d_info, h_info, sizeof(ImgInfo), hipMemcpyHostToDevice));

  const size_t chunkCount = ri.dataPlanes_.size();
  h_rawChunks = (anslib::AnsSymbolType **)malloc(chunkCount*sizeof(anslib::AnsSymbolType*));
  h_chunkCounts = (anslib::AnsCountsType **)malloc(chunkCount*sizeof(anslib::AnsCountsType*));
  h_chunkCumul = (anslib::AnsCountsType **)malloc(chunkCount*sizeof(anslib::AnsCountsType*));
  h_compChunks = (anslib::AnsCompType **)malloc(chunkCount*sizeof(anslib::AnsCompType*));

  h_rawChunks = (anslib::AnsSymbolType **)malloc(chunkCount * sizeof(rawChunks));
  for(size_t chunkIdx = 0; chunkIdx < chunkCount; ++chunkIdx) {
    // chunk sizes may slightly vary for the chunks located at the edges of a plane
    const size_t chunkSize = ri.dataPlanes_.at(chunkIdx).size() * sizeof(anslib::AnsSymbolType *);
    CHECK_CUDA_ERROR(hipMalloc(&h_rawChunks[chunkIdx], chunkSize * sizeof(anslib::AnsSymbolType)));
    CHECK_CUDA_ERROR(hipMalloc(&h_chunkCounts[chunkIdx], anslib::ansCountsSize * sizeof(anslib::AnsCountsType)));
    CHECK_CUDA_ERROR(hipMalloc(&h_chunkCumul[chunkIdx], anslib::ansCumulSize * sizeof(anslib::AnsCountsType)));
    CHECK_CUDA_ERROR(hipMalloc(&h_compChunks[chunkIdx], chunkSize * sizeof(anslib::AnsCompType)));

    CHECK_CUDA_ERROR(hipMemcpy(h_rawChunks[chunkIdx], ri.dataPlanes_.at(chunkIdx).data(),
      anslib::ansCountsSize, hipMemcpyHostToDevice));
  }

  CHECK_CUDA_ERROR(hipMalloc(&rawChunks, chunkCount*sizeof(void*)));
  CHECK_CUDA_ERROR(hipMalloc(&chunkCounts, chunkCount*sizeof(void*)));
  CHECK_CUDA_ERROR(hipMalloc(&chunkCumul, chunkCount*sizeof(void*)));
  CHECK_CUDA_ERROR(hipMalloc(&compChunks, chunkCount*sizeof(void*)));

  CHECK_CUDA_ERROR(hipMemcpy((void*)rawChunks, (void*)h_rawChunks, chunkCount*sizeof(void*), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy((void*)chunkCounts, (void*)h_chunkCounts, chunkCount*sizeof(void*), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy((void*)compChunks, (void*)h_compChunks, chunkCount*sizeof(void*), hipMemcpyHostToDevice));
}

ImageDev::ImageDev(const anslib::CompImage &ci) {
  printf("constructor called\n");
  
  CHECK_CUDA_ERROR(hipMalloc((void**)&d_info, sizeof(ImgInfo)));
  h_info = (ImgInfo *)malloc(sizeof(struct ImgInfo));
  ImgInfo tmpInfo = {ci.width_, ci.height_, ci.numOfPlanes_, ci.chunksPerPlaneCount(), ci.chunkWidth_};
  memcpy(h_info, &tmpInfo, sizeof(ImgInfo));
  CHECK_CUDA_ERROR(hipMemcpy(d_info, h_info, sizeof(ImgInfo), hipMemcpyHostToDevice));

  const size_t chunkCount = ci.compressedPlanes_.size();
  h_rawChunks = (anslib::AnsSymbolType **)malloc(chunkCount*sizeof(anslib::AnsSymbolType*));
  h_chunkCounts = (anslib::AnsCountsType **)malloc(chunkCount*sizeof(anslib::AnsCountsType*));
  h_chunkCumul = (anslib::AnsCountsType **)malloc(chunkCount*sizeof(anslib::AnsCountsType*));
  h_compChunks = (anslib::AnsCompType **)malloc(chunkCount*sizeof(anslib::AnsCompType*));

  h_rawChunks = (anslib::AnsSymbolType **)malloc(chunkCount * sizeof(rawChunks));
  for(size_t chunkIdx = 0; chunkIdx < chunkCount; ++chunkIdx) {
    // chunk sizes will vary for all compressed chunks
    const size_t chunkSize = ci.compressedPlanes_.at(chunkIdx).plane.size();

    CHECK_CUDA_ERROR(hipMalloc(&h_rawChunks[chunkIdx], chunkSize * sizeof(anslib::AnsSymbolType)));
    CHECK_CUDA_ERROR(hipMalloc(&h_chunkCounts[chunkIdx], anslib::ansCountsSize * sizeof(anslib::AnsCountsType)));
    CHECK_CUDA_ERROR(hipMalloc(&h_chunkCumul[chunkIdx], anslib::ansCumulSize * sizeof(anslib::AnsCountsType)));
    CHECK_CUDA_ERROR(hipMalloc(&h_compChunks[chunkIdx], chunkSize * sizeof(anslib::AnsCompType)));

    CHECK_CUDA_ERROR(hipMemcpy(h_chunkCounts[chunkIdx], ci.compressedPlanes_.at(chunkIdx).counts.data(),
      anslib::ansCountsSize, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(h_compChunks[chunkIdx], ci.compressedPlanes_.at(chunkIdx).plane.data(),
      chunkSize, hipMemcpyHostToDevice));
  }

  
  CHECK_CUDA_ERROR(hipMalloc(&rawChunks, chunkCount*sizeof(void*)));
  CHECK_CUDA_ERROR(hipMalloc(&chunkCounts, chunkCount*sizeof(void*)));
  CHECK_CUDA_ERROR(hipMalloc(&chunkCumul, chunkCount*sizeof(void*)));
  CHECK_CUDA_ERROR(hipMalloc(&compChunks, chunkCount*sizeof(void*)));

  CHECK_CUDA_ERROR(hipMemcpy(rawChunks, h_rawChunks, chunkCount*sizeof(void*), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(chunkCounts, h_chunkCounts, chunkCount*sizeof(void*), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(chunkCumul, h_chunkCumul, chunkCount*sizeof(void*), hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(compChunks, h_compChunks, chunkCount*sizeof(void*), hipMemcpyHostToDevice));
}

ImageDev::~ImageDev() {
  printf("destructor called\n");

  // release device-side chunk data arrays
  for(size_t chunkIdx = 0; chunkIdx < h_info->numOfChunksPerPlane_ * h_info->numOfPlanes_; ++chunkIdx) {
    if(h_rawChunks[chunkIdx])   hipFree(h_rawChunks[chunkIdx]);
    if(h_compChunks[chunkIdx])  hipFree(h_compChunks[chunkIdx]);
    if(h_chunkCounts[chunkIdx]) hipFree(h_chunkCounts[chunkIdx]);
    if(h_chunkCumul[chunkIdx])  hipFree(h_chunkCumul[chunkIdx]);
  }

  // release host-side pointer arrays
  if(h_rawChunks) free(h_rawChunks);
  if(h_compChunks) free(h_compChunks);
  if(h_chunkCounts) free(h_chunkCounts);
  if(h_chunkCumul) free(h_chunkCumul);
  
  // release device-side pointer arrays
  if(rawChunks) hipFree(rawChunks);
  if(compChunks) hipFree(compChunks);
  if(chunkCounts) hipFree(chunkCounts);
  if(chunkCumul) hipFree(chunkCumul);
  
  if(h_info) free(h_info);
  if(d_info) hipFree(d_info);
}


__device__ void countSymbolsInPlane(anslib::AnsSymbolType *symbols, anslib::AnsCountsType *counts) {
  printf("ayooo\n");
}
__global__ void g_compressImage(ImageDev *img) {
  countSymbolsInPlane(img->rawChunks[0], img->chunkCounts[0]);
}


hipDeviceProp_t printGPUproperties() {
  int nDevices;
  hipDeviceProp_t prop;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
          prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
          prop.memoryBusWidth);
    printf("  Concurrent Kernels: %d\n",
          prop.concurrentKernels);
    printf("  Async Engine Count: %d\n",
          prop.asyncEngineCount);      
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
          2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Shared Memory Per Block: %d\n",
          prop.sharedMemPerBlock);
    printf("  Multiprocessor Count: %d\n",
          prop.multiProcessorCount);
    printf("  Max Blocks Per Multiprocessor: %d\n",
          prop.maxBlocksPerMultiProcessor);
    printf("  Max Threads Per Block: %d\n",
          prop.maxThreadsPerBlock);
    printf("  Max Threads Per Multiprocessor: %d\n\n",
          prop.maxThreadsPerMultiProcessor);
  }
  return prop;
}

__global__ void g_splitIntoChunks_body(anslib::AnsSymbolType **inPlanes, anslib::AnsSymbolType **outChunks,
    size_t planeWid, size_t squareChunkWid, size_t numOfChunksPerPlane, size_t chunksCount_hor) {
  extern __shared__ anslib::AnsSymbolType sChunkLocal[];
  const int xOffsetOfChunk = blockIdx.x * squareChunkWid;
  const int yOffsetOfChunk = blockIdx.y * squareChunkWid;
  for (size_t rowOffset = 0; rowOffset + blockDim.y < squareChunkWid; rowOffset += blockDim.y) {
    const int scindex = (rowOffset + threadIdx.y) * squareChunkWid + threadIdx.x;
    const size_t inPlanePixIdx = (yOffsetOfChunk + rowOffset +
      threadIdx.y) * planeWid + xOffsetOfChunk + threadIdx.x;
    sChunkLocal[scindex] = inPlanes[blockIdx.z][inPlanePixIdx];
  }
  __syncthreads();
  if(threadIdx.x == 0 && threadIdx.y == 0) {
    const size_t outChunkIdx = blockIdx.z * numOfChunksPerPlane + blockIdx.y * chunksCount_hor + blockIdx.x;
    const size_t chunkSizeBytes = sizeof(anslib::AnsSymbolType)*squareChunkWid*squareChunkWid;
    // outChunks[outChunkIdx] = (anslib::AnsSymbolType *)malloc(chunkSizeBytes);
    //if(outChunks[outChunkIdx]) {
    memcpy(outChunks[outChunkIdx], sChunkLocal, chunkSizeBytes);
    //}
  }
  
}

/* If the image dimensions are not multiplicities of chunk size, right and bottom edge chunks contain
 more columns / rows - this kernel takes care of partitioning at right and bottom edge; 
 Consecutive blocks correspond to the image chunks in the following order:
|x |x |0  |
|x |x |1  |
|x |x |2  |
|4 |5 |3  |
|  |  |   |
*/
__global__ void g_splitIntoChunks_edges(anslib::AnsSymbolType **inPlanes, anslib::AnsSymbolType **outChunks,
    size_t planeHei, size_t planeWid, size_t squareChunkWid, size_t edgeChunkWid, size_t edgeChunkHei,
    size_t chunkCount_ver, size_t chunkCount_hor, bool isRightEdgeMisaligned, bool isBottomEdgeMisaligned) {
  extern __shared__ anslib::AnsSymbolType sChunkLocal[];
  const bool isRightEdge = (isRightEdgeMisaligned && isBottomEdgeMisaligned) ?
    (blockIdx.x < chunkCount_ver) : isRightEdgeMisaligned;
  const bool isBottomEdge = (isRightEdgeMisaligned && isBottomEdgeMisaligned) ?
    (blockIdx.x >= chunkCount_ver) : ! isRightEdgeMisaligned;
  
  // right and bottom edge chunks are numbered top to bottom, right to left
  const size_t chunkIdx_x = isRightEdge ? chunkCount_hor - 1 : blockIdx.x - chunkCount_ver;
  const size_t chunkIdx_y = isRightEdge ? blockIdx.x : chunkCount_ver - 1;
  
  const size_t xOffsetOfChunk = squareChunkWid * chunkIdx_x;
  const size_t yOffsetOfChunk = squareChunkWid * chunkIdx_y;

  const size_t chunkWid = isRightEdge ? edgeChunkWid : squareChunkWid;
  const size_t chunkHei = isBottomEdge ? edgeChunkHei : squareChunkWid;

  // because the threads are reserved for the maximal width of a chunk,
  // the thread number may exceed the chunk width - checking it here:
  if (threadIdx.x < chunkWid) { 
    for (size_t rowOffset = 0; rowOffset < chunkHei; rowOffset += blockDim.y) {
      const size_t scindex = (rowOffset + threadIdx.y) * chunkWid + threadIdx.x;
      const size_t inPlanePixIdx = (yOffsetOfChunk + rowOffset + threadIdx.y) *
        planeWid + xOffsetOfChunk + threadIdx.x;
      sChunkLocal[scindex] = inPlanes[blockIdx.z][inPlanePixIdx];
    }
  }

  __syncthreads();
  if(threadIdx.x == 0 && threadIdx.y == 0) {
    const size_t outChunkIdx = blockIdx.z * chunkCount_ver * chunkCount_hor + chunkCount_hor * chunkIdx_y + chunkIdx_x;
    const size_t chunkSizeBytes = sizeof(anslib::AnsSymbolType)*chunkWid*chunkHei;
    outChunks[outChunkIdx] = (anslib::AnsSymbolType *)malloc(chunkSizeBytes);
    memcpy(outChunks[outChunkIdx], sChunkLocal, chunkSizeBytes);
  }
  
}

void run_g_splitIntoChunks(ImageDev *img, size_t chunkWid) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  assert(prop.maxThreadsPerBlock > chunkWid);
  const size_t threadRows = prop.maxThreadsPerBlock / chunkWid;
  const size_t chunksCount_ver = img->h_info->height_ / chunkWid;
  const size_t chunksCount_hor = img->h_info->width_ / chunkWid;
  const size_t chunksCount = chunksCount_ver * chunksCount_hor;
  const size_t edgeChunkHei = img->h_info->height_ - chunksCount_ver * chunkWid + chunkWid;
  const size_t edgeChunkWid = img->h_info->width_ - chunksCount_hor * chunkWid + chunkWid;

  // anslib::AnsSymbolType **h_rawChunkedPlanes = nullptr;
  anslib::AnsSymbolType **d_rawChunkedPlanes = nullptr;
  anslib::AnsSymbolType **h_rawChunkedPlanes = nullptr;

  // alloc array of pointers to chunked data
  img->h_info->numOfChunksPerPlane_ = chunksCount_ver * chunksCount_hor;
  const size_t numOfChunksTotal = img->h_info->numOfPlanes_ * img->h_info->numOfChunksPerPlane_;
  const size_t chunkSizeBytes = chunkWid*chunkWid*sizeof(anslib::AnsSymbolType);
  
  CHECK_CUDA_ERROR(hipMalloc((void**)&d_rawChunkedPlanes, sizeof(anslib::AnsSymbolType*) * numOfChunksTotal));
  h_rawChunkedPlanes = (anslib::AnsSymbolType **)malloc(sizeof(anslib::AnsSymbolType*) * numOfChunksTotal);
  for (size_t y = 0; y < chunksCount_ver; ++y) {
    for (size_t x = 0; x < chunksCount_hor; ++x) {
      size_t chunkW = x == chunksCount_hor - 1 ? edgeChunkWid : chunkWid;
      size_t chunkH = x == chunksCount_ver - 1 ? edgeChunkHei : chunkWid;
      CHECK_CUDA_ERROR(hipMalloc((void**)&h_rawChunkedPlanes[y * chunksCount_hor + x], chunkH * chunkW * sizeof(anslib::AnsSymbolType)));
      hipMemset(h_rawChunkedPlanes[y * chunksCount_hor + x], 1, chunkH * chunkW * sizeof(anslib::AnsSymbolType));
    }
  }
  CHECK_CUDA_ERROR(hipMemcpy(d_rawChunkedPlanes, h_rawChunkedPlanes,
    sizeof(anslib::AnsSymbolType*) * numOfChunksTotal, hipMemcpyHostToDevice));
  //d_rawChunkedPlanes = h_rawChunkedPlanes;

  // execute chunking
  // 2D block indexes correspond to chunk location in plane
  dim3 grid(chunksCount_hor, chunksCount_ver, img->h_info->numOfPlanes_);
  // 2D thread indexes correspond to pixel location in chunk
  dim3 block(chunkWid, threadRows);
  g_splitIntoChunks_body<<<grid, block, chunkSizeBytes>>>(
    img->rawChunks, d_rawChunkedPlanes, img->h_info->width_, 
    chunkWid, img->h_info->numOfChunksPerPlane_, chunksCount_hor);

  const bool isHorMisaligned = img->h_info->width_ % chunkWid;
  const bool isVerMisaligned = img->h_info->height_ % chunkWid;
  if (isHorMisaligned || isVerMisaligned) {
    const dim3 edgeChunksBlockCount =
      (chunksCount_ver * isVerMisaligned + chunksCount_hor * chunksCount_ver - 1 * (isVerMisaligned && isHorMisaligned),
      1, img->h_info->numOfPlanes_);
    const dim3 edgeChunkThreads = (edgeChunkWid, prop.maxThreadsPerBlock / edgeChunkWid);
    const size_t edgeChunkMaxSizeBytes = edgeChunkHei * edgeChunkWid * sizeof(anslib::AnsSymbolType);
    g_splitIntoChunks_edges<<<edgeChunksBlockCount, edgeChunkThreads, edgeChunkMaxSizeBytes>>>(
      img->rawChunks, d_rawChunkedPlanes, img->h_info->height_, img->h_info->width_, chunkWid,
      edgeChunkWid, edgeChunkHei, chunksCount_ver, chunksCount_hor, isHorMisaligned, isVerMisaligned);
  }
  hipDeviceSynchronize();
  // free non-chunked data arrays pointed by rawChunks
  for (size_t i = 0; i < img->h_info->numOfPlanes_; ++i) {
    hipFree(img->h_rawChunks[i]);
  }
  hipFree(img->rawChunks);

  // swap rawChunks and h_rawChunks with chunked d_rawChunkedPlanes
  img->rawChunks = d_rawChunkedPlanes;
  img->h_rawChunks = d_rawChunkedPlanes;

  // set corresponding metadata
  img->h_info->chunkWidth_ = chunkWid;
  img->h_info->numOfChunksPerPlane_ = chunksCount;
}


int main(int argc, char *argv[]) {
  std::vector<FileStats> encodeStats;
  std::vector<std::string> testImgs;
  listAllImgsInDir(CMAKE_SOURCE_DIR "/test_images/PHOTO_CD_KODAK/BMP_IMAGES/",
                   ".bmp", testImgs);
  printGPUproperties();
  anslib::RawImage rawImg = FileStats::getTestImg(testImgs.back());
  ImageDev compContext(rawImg);
  run_g_splitIntoChunks(&compContext, 64);
  /*for (auto imgPath : testImgs) {
   anslib::RawImage rawImg = FileStats::getTestImg(imgPath);
    for (uint32_t chunk_size = 0; chunk_size <= 512; chunk_size += 64) {    
      /// TODO: parallelize
      // rawImg.splitIntoChunks(chunk_size);
      anslib::CompImage compImg;
      anslib::AnsEncoder::compressImage(rawImg, compImg);
      {
        ImageDev compContext(rawImg);
        g_compressImage<<<3, 12>>>(&compContext);
        ImageDev g_decompContext(compImg);
      }
      
      /// TODO: make a 
      // anslib::AnsEncoder::compressImage(img, resultImg);
      // anslib::AnsDecoder::decompressImage(resultImg, img);

      // std::cout << "Processing " << imgPath.substr(imgPath.rfind('/') + 1) << " for chunk_size = " << chunk_size << '\n';
      // for (size_t i = 0; i < img.dataPlanes_.size(); ++i) {
      //   assert(img.dataPlanes_.at(i).size() == imgRef.dataPlanes_.at(i).size());
      //   assert(img.dataPlanes_.at(i) == imgRef.dataPlanes_.at(i));
      // }
      
      // FileStats fs(img, imgPath.substr(imgPath.rfind('/') + 1));
      // encodeStats.push_back(fs);
      // std::cout << encodeStats.back();
    // }
  }*/
}