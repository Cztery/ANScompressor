#include <hip/hip_runtime.h>

#include <cstdio>
#include <iostream>
#include <vector>

#include "benchlib.h"
#include "bmplib.h"
#include "common_nv.cuh"
#include "common.h"
#include "compressor.h"
#include "image.cuh"
#include "image.h"
#include "ppmlib.h"

int main(int argc, char *argv[]) {
  std::vector<FileStats> encodeStats;
  std::vector<std::string> testImgs;
  listAllImgsInDir(CMAKE_SOURCE_DIR "/test_images/PHOTO_CD_KODAK/BMP_IMAGES/",
                   ".bmp", testImgs);
  // printGPUproperties();
  anslib::RawImage rawImg = FileStats::getTestImg(testImgs.back());
  ImageDev compContext(rawImg);
  compContext.splitIntoChunks(64);
  {
    auto p0 = compContext.getPlane(0);
    auto p1 = compContext.getPlane(1);
    auto p2 = compContext.getPlane(2);
    anslib::RawImage chunked(p0, p1, p2,
                              compContext.h_info->width_, compContext.h_info->height_);
    anslib::bmplib::BmpImage out(testImgs.back().c_str());
    out.data = chunked.getPlanesAsBmpData();
    out.bmpWrite("chunked.bmp");
  }
  compContext.joinChunks();
  {
    auto p0 = compContext.getPlane(0);
    auto p1 = compContext.getPlane(1);
    auto p2 = compContext.getPlane(2);
    anslib::RawImage rejoined(p0, p1, p2,
                              compContext.h_info->width_, compContext.h_info->height_);
    anslib::bmplib::BmpImage out(testImgs.back().c_str());
    out.data = rejoined.getPlanesAsBmpData();
    out.bmpWrite("rejoined.bmp");
  
    if (rejoined.dataPlanes_[0] != rawImg.dataPlanes_[0])
      return 1;
    if (rejoined.dataPlanes_[1] != rawImg.dataPlanes_[1])
      return 2;
    if (rejoined.dataPlanes_[2] != rawImg.dataPlanes_[2])
      return 3;
    return 5;
  }
  /*for (auto imgPath : testImgs) {
   anslib::RawImage rawImg = FileStats::getTestImg(imgPath);
    for (uint32_t chunk_size = 0; chunk_size <= 512; chunk_size += 64) {
      /// TODO: parallelize
      // rawImg.splitIntoChunks(chunk_size);
      anslib::CompImage compImg;
      anslib::AnsEncoder::compressImage(rawImg, compImg);
      {
        ImageDev compContext(rawImg);
        g_compressImage<<<3, 12>>>(&compContext);
        ImageDev g_decompContext(compImg);
      }

      /// TODO: make a
      // anslib::AnsEncoder::compressImage(img, resultImg);
      // anslib::AnsDecoder::decompressImage(resultImg, img);

      // std::cout << "Processing " << imgPath.substr(imgPath.rfind('/') + 1) <<
  " for chunk_size = " << chunk_size << '\n';
      // for (size_t i = 0; i < img.dataPlanes_.size(); ++i) {
      //   assert(img.dataPlanes_.at(i).size() ==
  imgRef.dataPlanes_.at(i).size());
      //   assert(img.dataPlanes_.at(i) == imgRef.dataPlanes_.at(i));
      // }

      // FileStats fs(img, imgPath.substr(imgPath.rfind('/') + 1));
      // encodeStats.push_back(fs);
      // std::cout << encodeStats.back();
    // }
  }*/
}